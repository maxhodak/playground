#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "MatrixMath.h"

__global__ void add_vectors(float *a, float *b, float *c, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) {
    c[idx] = a[idx] + b[idx];
  }
}