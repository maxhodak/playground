#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

__global__ void add_vectors(float *a, float *b, float *c, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) {
    c[idx] = a[idx] + b[idx];
  }
}

int main(void) {
  float *a_h, *a_d, *b_h, *b_d, *c_h, *c_d;
  const int N = 10;
  size_t size = N * sizeof(float);
  a_h = (float *)malloc(size);
  b_h = (float *)malloc(size);
  c_h = (float *)malloc(size);

  srand(time(NULL));

  hipMalloc((void **) &a_d, size);
  hipMalloc((void **) &b_d, size);
  hipMalloc((void **) &c_d, size);

  for (int i=0; i<N; i++) {
    a_h[i] = rand() / (float)RAND_MAX;
    b_h[i] = rand() / (float)RAND_MAX;
  }

  hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
  hipMemcpy(b_d, b_h, size, hipMemcpyHostToDevice);

  int block_size = 4;
  int n_blocks = N/block_size + (N % block_size == 0 ? 0 : 1);
  add_vectors<<<n_blocks, block_size>>> (a_d, b_d, c_d, N);

  hipMemcpy(c_h, c_d, sizeof(float)*N, hipMemcpyDeviceToHost);

  for (int i=0; i<N; i++) {
    printf("%d\t%f\t%f\t= %f\n", i, a_h[i], b_h[i], c_h[i]);
  }

  free(a_h);
  free(b_h);
  free(c_h);
  hipFree(a_d);
  hipFree(b_d);
  hipFree(c_d);
}